#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/sequence.h>

// Error checking macros
#define gpuCheck(stmt) \
    do { \
        hipError_t err = stmt; \
        if (err != hipSuccess) { \
            printf("ERROR. Failed to run stmt %s\n", #stmt); \
            break; \
        } \
    } while (0)

#define cublasCheck(stmt) \
    do { \
        hipblasStatus_t err = stmt; \
        if (err != HIPBLAS_STATUS_SUCCESS) { \
            printf("ERROR. Failed to run cuBLAS stmt %s\n", #stmt); \
            break; \
        } \
    } while (0)

#define cusparseCheck(stmt) \
    do { \
        hipsparseStatus_t err = stmt; \
        if (err != HIPSPARSE_STATUS_SUCCESS) { \
            printf("ERROR. Failed to run cuSPARSE stmt %s\n", #stmt); \
            break; \
        } \
    } while (0)

// Timing functions
struct timeval t_start, t_end;
void cputimer_start() {
    gettimeofday(&t_start, 0);
}

void cputimer_stop(const char* info) {
    gettimeofday(&t_end, 0);
    double time = (1000000.0 * (t_end.tv_sec - t_start.tv_sec) + t_end.tv_usec - t_start.tv_usec);
    printf("Timing - %s. \t\tElapsed %.0f microseconds \n", info, time);
}

// Initialize the sparse matrix needed for the heat time step
void matrixInit(double* A, int* ArowPtr, int* AcolIndx, int dimX, double alpha) {
    // Implementation as in your example...
}

int main(int argc, char **argv) {
    // Variable definitions as in your example...
    int device = 0;
    int dimX;  // Dimension of the metal rod
    int nsteps;  // Number of time steps to perform
    double alpha = 0.4;  // Diffusion coefficient
    double *temp;  // Array to store the final time step
    double *A;  // Sparse matrix A values in the CSR format
    int *ARowPtr;  // Sparse matrix A row pointers in the CSR format
    int *AColIndx;  // Sparse matrix A col values in the CSR format
    int nzv;  // Number of non-zero values in the sparse matrix
    double *tmp;  // Temporal array of dimX for computations
    size_t bufferSize = 0;  // Buffer size needed by some routines
    void *buffer = nullptr;  // Buffer used by some routines in the libraries
    int concurrentAccessQ;  // Check if concurrent access flag is set
    double zero = 0;  // Zero constant
    double one = 1;  // One constant
    double norm;  // Variable for norm values
    double error;  // Variable for storing the relative error
    double tempLeft = 200.;  // Left heat source applied to the rod
    double tempRight = 300.;  // Right heat source applied to the rod
    hipblasHandle_t cublasHandle;  // cuBLAS handle
    hipsparseHandle_t cusparseHandle;  // cuSPARSE handle

    // Read the command line arguments and print them...
    dimX = atoi(argv[1]);
    nsteps = atoi(argv[2]);
    printf("The X dimension of the grid is %d \n", dimX);
    printf("The number of time steps to perform is %d \n", nsteps);

    // Check for concurrent managed access...
    gpuCheck(hipDeviceGetAttribute(&concurrentAccessQ, hipDeviceAttributeConcurrentManagedAccess, device));

    // Allocate memory using Unified Memory for temp, tmp, and the sparse matrix...
    cputimer_start();
    gpuCheck(hipMallocManaged(&temp, dimX * sizeof(double)));
    gpuCheck(hipMallocManaged(&tmp, dimX * sizeof(double)));
    gpuCheck(hipMallocManaged(&A, nzv * sizeof(double)));
    gpuCheck(hipMallocManaged(&ARowPtr, (dimX + 1) * sizeof(int)));
    gpuCheck(hipMallocManaged(&AColIndx, nzv * sizeof(int)));
    cputimer_stop("Allocating device memory");

    // Prefetch data to the appropriate locations...
    if (concurrentAccessQ) {
        cputimer_start();
        gpuCheck(hipMemPrefetchAsync(A, nzv * sizeof(double), hipCpuDeviceId));
        gpuCheck(hipMemPrefetchAsync(ARowPtr, (dimX + 1) * sizeof(int), hipCpuDeviceId));
        gpuCheck(hipMemPrefetchAsync(AColIndx, nzv * sizeof(int), hipCpuDeviceId));
        cputimer_stop("Prefetching GPU memory to the host");
    }

    // Initialize the sparse matrix...
    cputimer_start();
    matrixInit(A, ARowPtr, AColIndx, dimX, alpha);
    cputimer_stop("Initializing the sparse matrix on the host");

    // Initialize the boundary conditions...
    cputimer_start();
    memset(temp, 0, sizeof(double) * dimX);
    temp[0] = tempLeft;
    temp[dimX - 1] = tempRight;
    cputimer_stop("Initializing memory on the host");

    // Create the cuBLAS and cuSPARSE handles...
    cublasCheck(hipblasCreate(&cublasHandle));
    cusparseCheck(hipsparseCreate(&cusparseHandle));

    // Set the cuBLAS pointer mode to HIPBLAS_POINTER_MODE_HOST...
    cublasCheck(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST));

    // Create the matrix descriptor and vector descriptors for temp and tmp...
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    cusparseCheck(hipsparseCreateCsr(&matA, dimX, dimX, nzv, ARowPtr, AColIndx, A, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    cusparseCheck(hipsparseCreateDnVec(&vecX, dimX, temp, HIP_R_64F));
    cusparseCheck(hipsparseCreateDnVec(&vecY, dimX, tmp, HIP_R_64F));

// Calculate buffer size and allocate buffer for cuSPARSE operations
cusparseCheck(hipsparseSpMV_bufferSize(
    cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, vecX, &zero, vecY, 
    HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
gpuCheck(hipMalloc(&buffer, bufferSize));

for (int it = 0; it < nsteps; ++it) {
    cusparseCheck(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, vecX, &zero, vecY, 
        HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    cublasCheck(hipblasDaxpy(cublasHandle, dimX, &alpha, tmp, 1, temp, 1));
    cublasCheck(hipblasDnrm2(cublasHandle, dimX, temp, 1, &norm));

    if (norm < 1e-4)
        break;
}
    // Calculate the exact solution using thrust...
    thrust::device_ptr<double> thrustPtr(tmp);
    thrust::sequence(thrustPtr, thrustPtr + dimX, tempLeft, (tempRight - tempLeft) / (dimX - 1));

    // Calculate the relative approximation error...
    one = -1;
    cublasCheck(hipblasDaxpy(cublasHandle, dimX, &one, temp, 1, tmp, 1));
    cublasCheck(hipblasDnrm2(cublasHandle, dimX, tmp, 1, &norm));
    error = norm;
    cublasCheck(hipblasDnrm2(cublasHandle, dimX, temp, 1, &norm));
    error /= norm;
    printf("The relative error of the approximation is %f\n", error);

    // Destroy the matrix descriptor and vector descriptors...
    cusparseCheck(hipsparseDestroySpMat(matA));
    cusparseCheck(hipsparseDestroyDnVec(vecX));
    cusparseCheck(hipsparseDestroyDnVec(vecY));

    // Destroy the cuSPARSE and cuBLAS handles...
    cusparseCheck(hipsparseDestroy(cusparseHandle));
    cublasCheck(hipblasDestroy(cublasHandle));

    // Deallocate memory...
    hipFree(temp);
    hipFree(tmp);
    hipFree(A);
    hipFree(ARowPtr);
    hipFree(AColIndx);
    hipFree(buffer);

    return 0;
}
